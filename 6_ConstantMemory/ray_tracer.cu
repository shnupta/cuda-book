#include "hip/hip_runtime.h"
#include <stdio.h>

#include "../common/cpu_bitmap.h"

#define INF 2e10f
#define rnd(x) (x * rand() / RAND_MAX)
#define SPHERES 20
#define DIM 1024

struct Sphere {
  float r, g, b;
  float radius;
  float x, y, z;

  __device__ float hit(float ox, float oy, float *n) {
    float dx = ox - x;
    float dy = oy - y;
    if (dx*dx + dy*dy < radius*radius) {
      float dz = sqrtf(radius*radius - dx*dx - dy*dy);
      *n = dz / sqrtf(radius * radius);
      return dz + z;
    }
    return -INF;
  }
};

__constant__ Sphere s[SPHERES];

__global__ void kernel(unsigned char *ptr) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  int offset = x + y * blockIdx.x * gridDim.x;
  float ox = (x - DIM / 2);
  float oy = (y - DIM / 2);

  float r = 0, g = 0, b = 0;
  float maxz = -INF;
  for (int i = 0; i < SPHERES; i++) {
    float n, t = s[i].hit(ox, oy, &n);
    if (t > maxz) {
      float fscale = n;
      r = s[i].r * fscale;
      g = s[i].g * fscale;
      b = s[i].b * fscale;
      maxz = t;
    }
  }

  ptr[offset * 4 + 0] = (int) (r * 255);
  ptr[offset * 4 + 1] = (int) (g * 255);
  ptr[offset * 4 + 2] = (int) (b * 255);
  ptr[offset * 4 + 3] = 255;
}

int main() {
  // Capture the start time
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  CPUBitmap bitmap(DIM, DIM);
  unsigned char *dev_bitmap;

  hipMalloc((void**)&dev_bitmap, bitmap.image_size());

  // Allocate temp memory, initialise, copy to GPU and free temp memory
  Sphere *temp_s = (Sphere *) malloc(sizeof(Sphere) * SPHERES);
  for (int i = 0; i < SPHERES; i++) {
    temp_s[i].r = rnd(1.0f);
    temp_s[i].g = rnd(1.0f);
    temp_s[i].b = rnd(1.0f);
    temp_s[i].x = rnd(1000.0f) - 500;
    temp_s[i].y = rnd(1000.0f) - 500;
    temp_s[i].z = rnd(1000.0f) - 500;
    temp_s[i].radius = rnd(100.0f) + 20;
  }

  hipMemcpyToSymbol(HIP_SYMBOL(s), temp_s, sizeof(Sphere) * SPHERES, hipMemcpyHostToDevice);
  free(temp_s);

  // Generate a bitmap from our sphere data
  dim3 grids(DIM / 16, DIM / 16);
  dim3 threads(16, 16);
  kernel<<<grids, threads>>>(dev_bitmap);

  // Copy bitmap back from GPU
  hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf("Time to generate: %3.1f ms\n", elapsedTime);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  bitmap.display_and_exit();

  hipFree(dev_bitmap);
  hipFree(s);

  return 0;
}
